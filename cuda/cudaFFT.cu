#include "hip/hip_runtime.h"
/**
 * @author HAN LI
 * @email  xxmlih05869@outlook.com
 * @time   2022.12.15
 */
#include<iostream>
#include<cmath>
#include<string>
#include<cstring>
#include <sys/time.h>
#include "support.h"
#define BLOCK_SIZE 1024
using namespace std;
const int maxn=1e6+5; 

struct Complex //complex
{
    double x,y;
    Complex()
    {
		x=0.0;
		y=0.0;
    }
    
    void set(double dx,double dy)
    {
    	x=dx;
		y=dy;
	}
	
    __device__
	Complex(double dx,double dy)
    {
        x=dx;
        y=dy;
    }
};

Complex operator +(Complex a,Complex b)
{
	Complex tmp;
	tmp.set(a.x+b.x,a.y+b.y);
    return tmp;
}
Complex operator -(Complex a,Complex b)
{
	Complex tmp;
	tmp.set(a.x-b.x,a.y-b.y);
    return tmp;
}
Complex operator *(Complex a,Complex b)
{
	Complex tmp;
	tmp.set(a.x*b.x-a.y*b.y,a.x*b.y+a.y*b.x);
    return tmp;
}

__device__
void gpuset(Complex& it,double dx=0,double dy=0)
{
	it.x=dx;
	it.y=dy;
}
__device__
Complex gpuadd(const Complex& a,const Complex& b)
{
	return Complex(a.x+b.x,a.y+b.y);
}

__device__
Complex gpusub(const Complex& a,const Complex& b)
{
	return Complex(a.x-b.x,a.y-b.y);
}

__device__
Complex gpumul(const Complex& a,const Complex& b)
{
	return Complex(a.x*b.x-a.y*b.y,a.x*b.y+a.y*b.x);
}

const double pi=acos(-1.0); //PI
int limit=1,bit=0; //limit is the final length extended. limit = 1<<bit
int wz[maxn<<2];
int re[maxn<<2]; //save results
Complex a[maxn<<2],b[maxn<<2];
char s1[maxn],s2[maxn];//save two input numbers

Timer timer;
Complex* da,*db,*dc;
int* dwz;

__global__
void FFT(Complex* __restrict__ a,int limit,int flag)
{
	for(int stride = 2; stride <= limit; stride <<= 1) 
	{
		const double _PI = acos(-1.0);
		int validx = (threadIdx.x + blockIdx.x * blockDim.x) * stride;
		for (int j = 0; j < (stride >> 1); j++) 
		{
			if (validx + j + (stride >> 1) < limit) 
			{
				Complex wn(cos(2.0*_PI*flag*j/stride), sin(2.0*_PI*flag*j/stride));
				Complex u = a[j + validx];
				Complex v = gpumul(wn, a[j + validx + stride / 2]);
				a[j + validx] = gpuadd(u, v);
				a[j + validx + (stride >> 1)] = gpusub(u, v);
			}
		}
		__syncthreads();
	}
}


__global__
void bitreverse(int* __restrict__ dwz, int bits, int limit){
	int i = (blockIdx.x * blockDim.x + threadIdx.x);
	int idx = i;
	int r = 0;
    do {
        r += i % 2 << --bits;
    } while (i /= 2);
    *(dwz+idx) = r;
}

int main(int argc,char* argv[])
{
    int returnflag = scanf("%s%s",s1,s2);
    int len1=strlen(s1),len2=strlen(s2);
    int len=len1+len2-2;//add times, denotes the highest order of the result polynominal
    len+=1;//n order polynominal needs n+1 points to represent it
    while(limit<len)//ensure that n is no less than the exponential power of 2
    {
        limit<<=1;
        bit++;
    }               
    //An n-digit decimal number can be viewed as an n-1 polynomial
    for(int i=len1-1,j=0;i>=0;i--,j++)
    {
        a[j].x=s1[i]-'0';
        a[j].y=0;
    }
    for(int i=len2-1,j=0;i>=0;i--,j++)
    {
        b[j].x=s2[i]-'0';
        b[j].y=0;
    }
    //for(int i=0;i<limit;i++)
    //    wz[i]=(wz[i>>1]>>1)|((i&1)<<(bit-1));
    const unsigned int numBlocks = ceil(1.0 * limit / BLOCK_SIZE);
    
    hipMalloc((void**)&dwz,sizeof(int) * limit);
    hipDeviceSynchronize();
    hipMemcpy(dwz, wz, sizeof(int) * (limit), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    startTime(&timer);
	bitreverse <<< numBlocks, BLOCK_SIZE >>> (dwz, bit, limit);
	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f\n", elapsedTime(timer));
    hipDeviceSynchronize();
    hipMemcpy(wz, dwz, sizeof(int) * (limit), hipMemcpyDeviceToHost);
    hipFree(dwz);
    
    
    for(int i=0;i<limit;i++)
        if(i<wz[i])
            swap(a[i],a[wz[i]]);
    for(int i=0;i<limit;i++)
        if(i<wz[i])
            swap(b[i],b[wz[i]]);     
	
    hipMalloc((void**)&da, sizeof(Complex) * (limit));
    hipMalloc((void**)&db, sizeof(Complex) * (limit));
    hipDeviceSynchronize();
    
    hipMemcpy(da, a, sizeof(Complex) * (limit), hipMemcpyHostToDevice);
    hipMemcpy(db, b, sizeof(Complex) * (limit), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    
   	
    startTime(&timer);
    FFT <<< numBlocks, BLOCK_SIZE >>> (da,limit,1);
	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f\n", elapsedTime(timer));
	hipMemcpy(a, da, sizeof(Complex) * (limit), hipMemcpyDeviceToHost);

    startTime(&timer);
	FFT <<< numBlocks, BLOCK_SIZE >>> (db,limit,1);
	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f\n", elapsedTime(timer));
	hipMemcpy(b, db, sizeof(Complex) * (limit), hipMemcpyDeviceToHost);
    
    for(int i=0;i<limit;i++)
        a[i]=a[i]*b[i];
    
    for(int i=0;i<limit;i++)
        if(i<wz[i])
            swap(a[i],a[wz[i]]);
    
    hipMemcpy(da, a, sizeof(Complex) * (limit), hipMemcpyHostToDevice);
    startTime(&timer);
	FFT <<< numBlocks, BLOCK_SIZE >>> (da,limit,-1);
	hipDeviceSynchronize();
	stopTime(&timer); 
	printf("%f\n", elapsedTime(timer));
	hipMemcpy(a, da, sizeof(Complex) * (limit), hipMemcpyDeviceToHost);

    memset(re,0,sizeof(re));
    for(int i=0;i<=limit;i++)
    {
        re[i]+=(int)(a[i].x/limit+0.5);
        if(re[i]>=10) //carry bit
        {
            re[i+1]+=re[i]/10;
            re[i]%=10;
            if(i==limit)
                ++limit;
        }
    }
    while(limit&&!re[limit])//exclude high order zero
        limit--;
    FILE* fp = fopen("./res","w");
    while(limit>=0)
        fprintf(fp,"%d",re[limit--]);
    fprintf(fp,"\n");
    fclose(fp);
    hipFree(da);
    hipFree(db);
    return 0;

}
