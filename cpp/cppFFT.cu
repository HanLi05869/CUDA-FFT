#include "hip/hip_runtime.h"
/**
 * @author HAN LI
 * @email  xxmlih05869@outlook.com
 * @time   2022.12.15
 */
#include<iostream>
#include<cmath>
#include<string>
#include<cstring>
#include <sys/time.h>
#include "support.h"
using namespace std;
const int maxn=1e6+5;
 
struct Complex //complex
{
    double x,y;
    Complex(double dx=0,double dy=0)
    {
        x=dx;
        y=dy;
    }
};

Complex operator +(Complex a,Complex b)
{
    return Complex(a.x+b.x,a.y+b.y);
}
Complex operator -(Complex a,Complex b)
{
    return Complex(a.x-b.x,a.y-b.y);
}
Complex operator *(Complex a,Complex b)
{
    return Complex(a.x*b.x-a.y*b.y,a.x*b.y+a.y*b.x);
}

const double pi=acos(-1.0); //PI
int limit=1,bit=0; //limit is the final length extended. limit = 1<<bit
int wz[maxn<<2];
int re[maxn<<2]; //save results
Complex a[maxn<<2],b[maxn<<2];
char s1[maxn],s2[maxn];//save two input numbers

void FFT(Complex *A,int inv)
{
    //base on the reverse bit sequience of the number,just swap the corresponding elements
    for(int mid=1;mid<limit;mid<<=1)
    {
        Complex wn(cos(pi/mid),inv*sin(pi/mid));
        for(int i=0;i<limit;i+=(mid<<1))
        {
            Complex w(1,0);
            for(int j=0;j<mid;j++,w=w*wn)
            {
                Complex t1=A[i+j];
                Complex t2=w*A[i+mid+j];
                A[i+j]=t1+t2;
                A[i+mid+j]=t1-t2;
            }
        }
    }
}

Timer timer;
int main(int argc,char* argv[])
{
    int returnflg = scanf("%s%s",s1,s2);
    int len1=strlen(s1),len2=strlen(s2);
    int len=len1+len2-2;//add times, denotes the highest order of the result polynominal
    len+=1;//n order polynominal needs n+1 points to represent it
    while(limit<len)//ensure that n is no less than the exponential power of 2
    {
        limit<<=1;
        bit++;
    }               
    //An n-digit decimal number can be viewed as an n-1 polynomial
    for(int i=len1-1,j=0;i>=0;i--,j++)
    {
        a[j].x=s1[i]-'0';
        a[j].y=0;
    }
    for(int i=len2-1,j=0;i>=0;i--,j++)
    {
        b[j].x=s2[i]-'0';
        b[j].y=0;
    }
    startTime(&timer);
    for(int i=0;i<limit;i++)
        wz[i]=(wz[i>>1]>>1)|((i&1)<<(bit-1));//uncanny
    stopTime(&timer);
    printf("%f\n",elapsedTime(timer)); fflush(stdout);
    
    for(int i=0;i<limit;i++)
        if(i<wz[i])
            swap(a[i],a[wz[i]]);
    startTime(&timer);
    FFT(a,1);
    stopTime(&timer);
    printf("%f\n",elapsedTime(timer)); fflush(stdout);

    for(int i=0;i<limit;i++)
        if(i<wz[i])
            swap(b[i],b[wz[i]]);
    startTime(&timer);
    FFT(b,1);
    stopTime(&timer);
    printf("%f\n",elapsedTime(timer)); fflush(stdout);
    
    for(int i=0;i<limit;i++)
        a[i]=a[i]*b[i];
    
    for(int i=0;i<limit;i++)
        if(i<wz[i])
            swap(a[i],a[wz[i]]);
    startTime(&timer);
    FFT(a,-1);
    stopTime(&timer);
    printf("%f\n",elapsedTime(timer)); fflush(stdout);
    memset(re,0,sizeof(re));
    for(int i=0;i<=limit;i++)
    {
        re[i]+=(int)(a[i].x/limit+0.5);
        if(re[i]>=10) //carry bit
        {
            re[i+1]+=re[i]/10;
            re[i]%=10;
            if(i==limit)
                ++limit;
        }
    }
    while(limit&&!re[limit])//exclude high order zero
        limit--;
    FILE* fp = fopen("./res","w");
    while(limit>=0)
        fprintf(fp,"%d",re[limit--]);
    fprintf(fp,"\n");
    fclose(fp);
    return 0;
}
